#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>

#include "bmp_class.h"

class Complex_double
{
public:
	double real;
	double img;
public:
	Complex_double()
	{
		real = 0;
		img = 0;
	}
	Complex_double(double a1,double a2)
	{
		real = a1;
		img = a2;
	}
	friend Complex_double operator*(const Complex_double&,const Complex_double&);
	friend Complex_double operator+(const Complex_double&,const Complex_double&);
	friend Complex_double operator-(const Complex_double&,const Complex_double&);

	double modsize()
	{
		return sqrt(real*real + img*img);
	}
};

Complex_double operator*(const Complex_double&a1,const Complex_double&a2)
{
	return Complex_double(a1.real*a2.real-a1.img*a2.img,a1.real*a2.img+a1.img+a2.real);
}
Complex_double operator+(const Complex_double&a1,const Complex_double&a2)
{
	return Complex_double(a1.real+a2.real,a1.img+a2.img);
}
Complex_double operator-(const Complex_double&a1,const Complex_double&a2)
{
	return Complex_double(a1.real-a2.real,a1.img-a2.img);
}
void print_usage()
{
	printf("usage:./julia width height real img file_name\n");
	exit(1);
}
__device__ int julia_map(int julia_n)
{

	int B,G,R;
	int AB = 0x1,BB = 0x12;
	int AG = 0x12,BG = 0x2;
	int AR = 0x4,BR = 0x20;

	B = AB*julia_n + BB;
	if(B > 0x0ff)B = 0x0ff;
	G = AG*julia_n + BG;
	if(G > 0x0ff)G = 0x0ff;
	R = AR*julia_n + BR;
	if(R > 0x0ff)R = 0x0ff;

	return B*0x010000+G*0x0100+R;
}

__global__ void julia_compute(int*n_tab,int img_width,int img_height,double c_real,double c_img)
{
	int tid = blockDim.x*blockIdx.x + threadIdx.x;
	//printf("tid:%d\n",tid);
	int x_id = tid%img_width;
	int y_id = tid/img_height;

	//compute

	//julia
	double zn_real = ((double)x_id/img_height-0.5)*2-0.74;
	double zn_img = ((double)y_id/img_width-0.5)*3;

	//mandelbrot
	/*
	c_real = (double)x_id/img_height-0.5;
	c_img = (double)y_id/img_width-0.5;
	double zn_real = 0;
	double zn_img = 0;
	*/
	if(tid < img_height*img_width)
	{
		int MAX_N = 2000;

		double MAX_MOD = 2.0;
		MAX_MOD = MAX_MOD*MAX_MOD;

		int n = 0; 
		for(n = 0;n < MAX_N;n++)
		{
			//Julia
			double tmp_real = zn_real*zn_real - zn_img*zn_img + c_real;
			double tmp_img = zn_img*zn_real*2 + c_img;
			if((tmp_real*tmp_real + tmp_img*tmp_img) > MAX_MOD)break;
			zn_real = tmp_real;
			zn_img = tmp_img;

		}
		n_tab[tid] = julia_map(n);
	}
}



void gpu_julia_compute(bmp_class &julia_ins,int img_width,int img_height,Complex_double c_const)
{
	//malloc cpu output
	int *cpu_output;
	cpu_output = (int*)malloc(sizeof(int)*img_width*img_height);
	if(cpu_output == NULL)
	{
		exit(-1);
	}
	//malloc gpu output
	int *gpu_output;
	hipMalloc((void**)&gpu_output,sizeof(int)*img_width*img_height);

	//launch kernel
	int BLOCK_SIZE = 512;
	int block_num = (img_width*img_height+BLOCK_SIZE-1)/BLOCK_SIZE;
	julia_compute<<<block_num,BLOCK_SIZE>>>(gpu_output,img_width,img_height,c_const.real,c_const.img);

	//mov data back
	hipMemcpy((void*)cpu_output,(const void*)gpu_output,sizeof(int)*img_width*img_height,hipMemcpyDeviceToHost);

	//write to julia set
	int count = 0;
	for(int i = 0;i < img_width;i++)
	{
		for(int j = 0;j < img_height;j++)
		{

			//if(cpu_output[j*img_width+i] >= 2000)count++;
			int color_tmp = cpu_output[j*img_width+i];
			julia_ins.datatab[j][i] = color_tmp;
		}
	}
	printf("Overflow ratio:%.3f\n",(count+0.0)/img_width/img_height);
	hipFree(gpu_output);

}

int main(int argc,char**argv)
{
	int img_width = 600;
	int img_height = 400;

	if(argc != 6)print_usage();
	img_width = atoi(argv[1]);
	img_height = atoi(argv[2]);
	double c_real = atof(argv[3]);
	double c_img = atof(argv[4]);

	Complex_double c_const(c_real, c_img);
	bmp_class julia_set;
	julia_set.resize(img_width,img_height);

	//double MAX = 200000.0;
	//double MAX_n = 2000;
	/*
	//cpu code
	for(int i = 0;i < img_height;i++)
	{
		for(int j = 0;j < img_width;j++)
		{
			double x_idx = (double)i/img_height;
			double y_idx = (double)j/img_width;
			Complex_double tmp(x_idx,y_idx);
			int n;
			for(n = 0;n < MAX_n;n++)
			{
				tmp = tmp*tmp-c_const;
				if(tmp.modsize() > MAX)break;	
			}

			printf("i:%3d,j:%3d,n:%4d\n",i,j,n);
			if(n <= MAX_n/3)julia_set.datatab[i][j] = 0x00ffff;
			else julia_set.datatab[i][j] = 0;
		}
	}
	*/

	//gpu version
	for(long i = 0;i < 20000000000;i++)
	{
		gpu_julia_compute(julia_set,img_width,img_height,c_const);
	}
	//julia_set.bmp_writedown(argv[5]);
	return 0;
}